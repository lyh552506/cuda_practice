#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include "../3rdparty/cutlass/include/cute/tensor.hpp"

#define PRINT(name, content) \
  print(name);               \
  print(" : ");              \
  print(content);            \
  print("\n");

#define PRINTTENSOR(name, content) \
  print(name);                     \
  print(" : ");                    \
  print_tensor(content);           \
  print("\n");

// #define PRINT_INFO
using namespace cute;

template <typename T>
void gen_rand_data(T *data, int n);

template <typename T, int kTileM, int kTileN, int kTileK, typename TiledMMA>
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n,
                            int k) {
  Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k),
                         make_stride(k, Int<1>{}));
  Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k),
                         make_stride(k, Int<1>{}));
  Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n),
                         make_stride(n, Int<1>{}));

  int ix = blockIdx.x;
  int iy = blockIdx.y;

  //  gA(kTileM, kTileK, num_tile_k)
  //  gB(kTileN, kTileK, num_tile_k)
  //  gC(kTileM, kTileN)
  Tensor gA =
      local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
  Tensor gB =
      local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
  Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}),
                         make_coord(iy, ix));

  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(threadIdx.x);
  // MMA_M = M / (mma_op_m * thr_layout_m)
  // MMA_N = N / (mma_op_n * thr_layout_n)
  // MMA_K = K / (mma_op_k * thr_layout_k)
  auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
  auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
  auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

  // register tensor, shape is different with global memory tensor
  auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
  auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
  auto tCrC = thr_mma.partition_fragment_C(gC(_, _));     // (MMA, MMA_M, MMA_N)
  clear(tCrC);

  // #ifdef PRINT_INFO
  /*
      gA : (_128,_32,32)
      tAgA : ((_2,_2,_2),_4,_2,32)
      tArA : ((_2,_2,_2),_4,_2)
      gB : (_128,_32,32)
      tBgB : ((_2,_2),_8,_2,32)
      tBrB : ((_2,_2),_8,_2)
      gC : (_128,_128)
      tCgC : ((_2,_2),_4,_8)
      tCrC : ((_2,_2),_4,_8)
  */
  if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
    PRINT("gA", gA.shape())
    PRINT("tAgA", tAgA.shape())
    PRINT("tArA", tArA.shape())

    PRINT("gB", gB.shape())
    PRINT("tBgB", tBgB.shape())
    PRINT("tBrB", tBrB.shape())

    PRINT("gC", gC.shape())
    PRINT("tCgC", tCgC.shape())
    PRINT("tCrC", tCrC.shape())
  }
  // #endif

  int num_tile_k = size<2>(gA);
#pragma unroll 1
  for (int itile = 0; itile < num_tile_k; ++itile) {
    // global memory to register
    // just use cute::copy, not tiled
    copy(tAgA(_, _, _, itile), tArA);
    copy(tBgB(_, _, _, itile), tBrB);

    // warp level, use  tiled_mma
    gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
  }
  // register to global memory
  copy(tCrC, tCgC);
}

int main() {
  srand(1000);

  using T = cute::half_t;
  hipEvent_t start, end;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&end);

  T *Cptr;
  T *Aptr;
  T *Bptr;

  int m = 1024 * 64;
  int n = 128;
  int k = 1024;

  //   hipMalloc(&Cptr, sizeof(T) * m * n);
  //   hipMalloc(&Aptr, sizeof(T) * m * k);
  //   hipMalloc(&Bptr, sizeof(T) * k * n);
  hipError_t err;
  err = hipMalloc(&Cptr, sizeof(T) * m * n);
  if (err != hipSuccess) {
    printf("hipMalloc failed for Cptr: %s\n", hipGetErrorString(err));
    return -1;
  }
  err = hipMalloc(&Aptr, sizeof(T) * m * k);
  if (err != hipSuccess) {
    printf("hipMalloc failed for Aptr: %s\n", hipGetErrorString(err));
    return -1;
  }
  err = hipMalloc(&Bptr, sizeof(T) * k * n);
  if (err != hipSuccess) {
    printf("hipMalloc failed for Bptr: %s\n", hipGetErrorString(err));
    return -1;
  }

  T *Aptr_host;
  T *Bptr_host;
  Aptr_host = (T *)malloc(sizeof(T) * m * k);
  Bptr_host = (T *)malloc(sizeof(T) * n * k);
  gen_rand_data(Aptr_host, m * k);
  gen_rand_data(Bptr_host, n * k);
  
  err =hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("hipMalloc failed for Bptr: %s\n", hipGetErrorString(err));
    return -1;
  }
  err =hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("hipMalloc failed for Bptr: %s\n", hipGetErrorString(err));
    return -1;
  }
  // M=16,N=8,K=16
  using mma_op = SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;
  using MMA = decltype(make_tiled_mma(
      mma_atom{},
      make_layout(Shape<_2, _2, _1>{}),    // the best shape for perf, use 2x2=4
                                           // warps = 128 threads
      make_layout(Shape<_1, _1, _1>{})));  // influence little to perf

  constexpr int kTileM = 128;
  constexpr int kTileN = 128;
  constexpr int kTileK = 32;

  // each thread block handle with (kTileM, kTileN) output
  dim3 grid(n / kTileN, m / kTileM);
  dim3 block(size(MMA{}));

  int count = 100;
  hipEventRecord(start);
  //   for (int i = 0; i < count; ++i) {
  gemm_simple<T, kTileM, kTileN, kTileK, MMA>
      <<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  //   }
  err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsedTime, start, end);
  std::cout << "gemm-simple took " << elapsedTime / count << "ms." << std::endl;
}

template <typename T>
void gen_rand_data(T *data, int n) {
  for (int i = 0; i < n; ++i) {
    float v = (rand() % 200 - 100) * 0.01;
    data[i] = v;
  }
}