#include "hip/hip_runtime.h"
#include "../3rdparty/ThunderKittens/include/kittens.cuh"
#include <iostream>
#include <string>
#include <fstream>

constexpr int ATTN_B = 16;
constexpr int ATTN_H = 16;
constexpr int ATTN_N = 1024; 
constexpr int ATTN_D = 128;
constexpr int ITER   = 10;

using namespace kittens;

constexpr int NUM_WORKERS = 4;
constexpr int PIPE_STAGES = 3; 

template<int D> constexpr size_t ROWS = 16*(128/D); // height of each worker tile (rows)
template<int D, typename T=bf16, typename L=row_l> using qkvo_tile = rt<T, ROWS<D>, D, L>;
template<int D, typename T=float> using attn_tile = rt<T, ROWS<D>, ROWS<D>>;
template<int D> using shared_tile = st_bf<ROWS<D>, D>;
template<int D> using global_layout = gl<bf16, -1, -1, -1, D>; // B, N, H, specified at runtime, D known at compile time for this kernel
template<int D> struct globals { global_layout<D> Qg, Kg, Vg, Og; };

template<int D> __launch_bounds__(NUM_WORKERS*WARP_THREADS, 1)
__global__ void attend_ker(const __grid_constant__ globals<D> g) {
    
    using load_group = kittens::group<2>; // pairs of workers collaboratively load k, v tiles
    int loadid = load_group::groupid(), workerid = kittens::warpid(); // which worker am I?
    constexpr int LOAD_BLOCKS = NUM_WORKERS / load_group::GROUP_WARPS;
    const int batch = blockIdx.z, head = blockIdx.y, q_seq = blockIdx.x * NUM_WORKERS + workerid;

    extern __shared__ alignment_dummy __shm[]; 
    shared_allocator al((int*)&__shm[0]);
    
    shared_tile<D> (&k_smem)[LOAD_BLOCKS][PIPE_STAGES] = al.allocate<shared_tile<D>, LOAD_BLOCKS, PIPE_STAGES>();
    shared_tile<D> (&v_smem)[LOAD_BLOCKS][PIPE_STAGES] = al.allocate<shared_tile<D>, LOAD_BLOCKS, PIPE_STAGES>();
    
    shared_tile<D> (&qo_smem)[NUM_WORKERS] = reinterpret_cast<shared_tile<D>(&)[NUM_WORKERS]>(k_smem);
    // Initialize all of the register tiles.
    qkvo_tile<D, bf16> q_reg, k_reg; // Q and K are both row layout, as we use mma_ABt.
    qkvo_tile<D, bf16, col_l> v_reg; // V is column layout, as we use mma_AB.
    qkvo_tile<D, float> o_reg; // Output tile.
    attn_tile<D, float> att_block; // attention tile, in float. (We want to use float wherever possible.)
    attn_tile<D, bf16> att_block_mma; // bf16 attention tile for the second mma_AB. We cast right before that op.
    typename attn_tile<D, float>::col_vec max_vec_last, max_vec, norm_vec; // these are column vectors for the in-place softmax.
    // each warp loads its own Q tile of 16x64
    if (q_seq*ROWS<D> < g.Qg.depth) {
        load<1, false>(qo_smem[workerid], g.Qg, {batch, q_seq, head, 0});  // going through shared memory improves coalescing of dram reads.
        __syncwarp();
        load(q_reg, qo_smem[workerid]);
    }
    __syncthreads();

    if constexpr(D == 64) mul(q_reg, q_reg, __float2bfloat16(0.125f * 1.44269504089));
    else if constexpr(D == 128) mul(q_reg, q_reg, __float2bfloat16(0.08838834764f * 1.44269504089));

    neg_infty(max_vec);
    zero(norm_vec);
    zero(o_reg);
    // launch the load of the first k, v tiles
    int kv_blocks = (g.Kg.depth + LOAD_BLOCKS*ROWS<D>-1) / (LOAD_BLOCKS*ROWS<D>), tic = 0;
    load_group::load_async<1, false>(k_smem[loadid][0], g.Kg, {batch, loadid, head, 0});
    load_group::load_async<1, false>(v_smem[loadid][0], g.Vg, {batch, loadid, head, 0});
    // iterate over k, v for these q's that have been loaded
    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic=(tic+1)%3) {
        int next_load_idx = (kv_idx+1)*LOAD_BLOCKS + loadid;
        if(next_load_idx*ROWS<D> < g.Kg.depth) {
            int next_tic = (tic+1)%3;
            load_group::load_async<1, false>(k_smem[loadid][next_tic], g.Kg, {batch, next_load_idx, head, 0});
            load_group::load_async<1, false>(v_smem[loadid][next_tic], g.Vg, {batch, next_load_idx, head, 0});
            load_async_wait<1>(); // next k, v can stay in flight.
        }
        else load_async_wait();
        __syncthreads();

        #pragma unroll LOAD_BLOCKS
        for(int subtile = 0; subtile < LOAD_BLOCKS && (kv_idx*LOAD_BLOCKS + subtile)*ROWS<D> < g.Kg.depth; subtile++) {
            load(k_reg, k_smem[subtile][tic]); // load k from shared into registers
            zero(att_block); // zero 16x16 attention tile
            mma_ABt(att_block, q_reg, k_reg, att_block); // Q@K.T
            int first_index = (kv_idx*LOAD_BLOCKS + subtile)*ROWS<D>; // one past the last KV index of this tile
            int start_fill = g.Kg.depth-first_index < ROWS<D> ? g.Kg.depth-first_index : ROWS<D>;
            right_fill(att_block, att_block, start_fill, base_types::constants<float>::neg_infty());
            copy(max_vec_last,  max_vec);
            row_max(max_vec, att_block, max_vec); 
            sub_row(att_block, att_block, max_vec); 
            exp2(att_block, att_block); 
            sub(max_vec_last, max_vec_last, max_vec); 
            exp2(max_vec_last, max_vec_last); 
            mul(norm_vec, norm_vec, max_vec_last); 
            row_sum(norm_vec, att_block, norm_vec); 
            copy(att_block_mma, att_block); 
            
            load(v_reg, v_smem[subtile][tic]); 
            mul_row(o_reg, o_reg, max_vec_last); 
            mma_AB(o_reg, att_block_mma, v_reg, o_reg);
        }
    }

    div_row(o_reg, o_reg, norm_vec);
    __syncthreads();
    if (q_seq*ROWS<D> < g.Og.depth) { // write out o.
        store(qo_smem[workerid], o_reg); // going through shared memory improves coalescing of dram writes.
        __syncwarp();
        store<1, false>(g.Og, qo_smem[workerid], {batch, q_seq, head, 0});
    }
}



#define BLOCK_SIZE (32*NUM_WORKERS)

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaCheckError( const char *file, const int line ) {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
}

// Compute FLOPs for forward attention
constexpr uint64_t ATTN_FLOPS = 
    2llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N * ATTN_D + // Q * K^T: 2BHNND (multiply-add)
    4llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N +          // Softmax: 2BHNN (exp and divide, plus flash-attn bookkeeping)
    2llu * ATTN_B * ATTN_H * ATTN_N * ATTN_N * ATTN_D;      // (Q * K^T) * V: 2BHNND (multiply-add)

int main(int argc, char **argv) {
    // TODO: consider doing sequential kernel launches to force batches dimension element to execute sequentially,
    // which may increase the probability of L2 cache hits on KV

    std::cout << "Entered main!" << std::endl;

    // create dummy variables that are the right size
    constexpr int TOTAL_ELEMENTS = ATTN_B*ATTN_H*ATTN_N*ATTN_D;
    constexpr int TOTAL_UNIQUE_ELEMENTS = ATTN_H*ATTN_N*ATTN_D;

    float *q = new float[TOTAL_ELEMENTS];
    float *k = new float[TOTAL_ELEMENTS];
    float *v = new float[TOTAL_ELEMENTS];
    float *o_ref = new float[TOTAL_ELEMENTS];

    bf16 *q_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *k_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *v_bf = new bf16[TOTAL_ELEMENTS];
    bf16 *o_bf = new bf16[TOTAL_ELEMENTS];
    float *o = new float[TOTAL_ELEMENTS];

    std::ifstream infile(argv[1]);

    std::cout << "Starting to enter!" << std::endl;

    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> q[i];
    std::cout << "Finished loading Q" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> k[i];
    std::cout << "Finished loading K" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> v[i];
    std::cout << "Finished loading V" << std::endl;
    for(int i = 0; i < TOTAL_ELEMENTS/ATTN_B; i++) infile >> o_ref[i];
    std::cout << "Finished loading O_REF" << std::endl;

    std::cout << "Finished loading file from " << argv[1] << "!" << std::endl;

    // replicate into batch elements
    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        q_bf[i] = __float2bfloat16(q[i % (TOTAL_ELEMENTS/ATTN_B)]);
        k_bf[i] = __float2bfloat16(k[i % (TOTAL_ELEMENTS/ATTN_B)]);
        v_bf[i] = __float2bfloat16(v[i % (TOTAL_ELEMENTS/ATTN_B)]);
    }

    bf16 *d_q, *d_k, *d_v, *d_o;
    hipMalloc(&d_q, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_k, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_v, TOTAL_ELEMENTS * sizeof(bf16));
    hipMalloc(&d_o, TOTAL_ELEMENTS * sizeof(bf16));

    hipMemcpy(d_q, q_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_k, k_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v_bf, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyHostToDevice);

    global_layout<ATTN_D> Qg(d_q, ATTN_B, ATTN_N, ATTN_H, nullptr);
    global_layout<ATTN_D> Kg(d_k, ATTN_B, ATTN_N, ATTN_H, nullptr);
    global_layout<ATTN_D> Vg(d_v, ATTN_B, ATTN_N, ATTN_H, nullptr);
    global_layout<ATTN_D> Og(d_o, ATTN_B, ATTN_N, ATTN_H, nullptr);
    globals<ATTN_D> g(Qg, Kg, Vg, Og);
    
    unsigned long mem_size = (kittens::MAX_SHARED_MEMORY) / 2; // have the flag tell us
    std::cout << "Max shared memory size: " << mem_size << std::endl;
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        attend_ker<ATTN_D>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    hipDeviceSynchronize();
    std::cout << "Starting kernel\n";
    dim3 grid((ATTN_N + qkvo_tile<ATTN_D>::rows*NUM_WORKERS - 1) / (qkvo_tile<ATTN_D>::rows*NUM_WORKERS), ATTN_H, ATTN_B);
    const auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < ITER; i++) {
        attend_ker<ATTN_D><<<grid, BLOCK_SIZE, mem_size>>>(g);
    }
    hipDeviceSynchronize();
    const auto finish = std::chrono::high_resolution_clock::now();
    CudaCheckError();
    std::cout << "Finished kernel\n";

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    // check correctness
    hipMemcpy(o_bf, d_o, TOTAL_ELEMENTS * sizeof(bf16), hipMemcpyDeviceToHost);
    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        o[i] = __bfloat162float(o_bf[i]);
    }

    bool good = true;
    float total_diff = 0;
    std::ofstream o_ref_file("printouts/o_ref.txt");
    std::ofstream o_file("printouts/o.txt");
    std::ofstream diff_file("printouts/diff.txt");

    total_diff = 0;

    for(int i = 0; i < TOTAL_ELEMENTS; i++) {
        float diff = o[i] - o_ref[i % (TOTAL_ELEMENTS/ATTN_B)];
        if(i < TOTAL_UNIQUE_ELEMENTS) {
            o_ref_file << o_ref[i % (TOTAL_ELEMENTS/ATTN_B)] << ' ';
            o_file << o[i] << ' ';
            diff_file << diff << ' ';
        }
        if(i % ATTN_D == ATTN_D-1) {
            o_ref_file << '\n';
            o_file << '\n';
            diff_file << '\n';
        }
        if(abs(diff) > 0.01 || isnan(diff)) {
            good = false;
        }
        total_diff += abs(diff);
    }
    std::cout << "Average diff: " << total_diff / TOTAL_UNIQUE_ELEMENTS << std::endl;
    std::cout << "Average execution time: " << std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count() / ITER << " us" << std::endl;
    if(good) std::cout << "Correct :)\n";
    else std::cout << "Incorrect :(\n";
    // Compute and print average TFLOPs achieved
    double avg_time_s = (double)(std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count()) / (ITER * 1e6);
    double avg_tflops = (ATTN_FLOPS / avg_time_s) / 1e12;
    std::cout << "Efficiency: " << avg_tflops << " TFLOPS\n\n\n" << std::endl;

    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);

    delete[] q, k, v, o, o_ref;
    delete[] q_bf, k_bf, v_bf, o_bf;

    return 0;
}