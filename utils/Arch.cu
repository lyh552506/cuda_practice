#include <hip/hip_runtime.h>

#include <iostream>

int main() {
  int device;
  hipDeviceProp_t prop;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
    std::cout << "  CUDA Capability (SM): " << deviceProp.major << "."
              << deviceProp.minor << std::endl;
  }

  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);

  // 标准的每个 Block 的共享内存大小
  std::cout << "Max shared memory per block: " << prop.sharedMemPerBlock
            << " bytes" << std::endl;

  // 查询支持的共享内存大小（可选分配）
  int sharedMemOptin = 0;
  hipDeviceGetAttribute(&sharedMemOptin,
                         hipDeviceAttributeSharedMemPerBlockOptin, device);
  std::cout << "Max opt-in shared memory per block: " << sharedMemOptin
            << " bytes" << std::endl;

  return 0;
}
