
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_vector_types.h>

template <typename tp>
int Performance(void (*kernel)(tp *, tp *, tp *, const int, const int,
                               const int),
                dim3 Grid, dim3 Block, int M, int N, int K, int repeat) {
  tp *d_a, *d_b, *d_c;
  hipMalloc(&d_a, sizeof(tp) * M * K);
  hipMalloc(&d_b, sizeof(tp) * K * N);
  hipMalloc(&d_c, sizeof(tp) * M * N);

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
  hipEventRecord(beg);
  for (int i = 0; i < repeat; i++)
    kernel<<<Grid, Block>>>(d_a, d_b, d_c, M, N, K);
  hipEventRecord(end);
  hipEventSynchronize(end);
  float msec, sec;
  hipEventElapsedTime(&msec, beg, end);
  sec = msec / 1000.0 / repeat;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return sec;
}