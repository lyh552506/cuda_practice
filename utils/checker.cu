#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int device;
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    // 标准的每个 Block 的共享内存大小
    std::cout << "Max shared memory per block: " 
              << prop.sharedMemPerBlock << " bytes" << std::endl;

    // 查询支持的共享内存大小（可选分配）
    int sharedMemOptin = 0;
    hipDeviceGetAttribute(&sharedMemOptin, hipDeviceAttributeSharedMemPerBlockOptin, device);
    std::cout << "Max opt-in shared memory per block: " 
              << sharedMemOptin << " bytes" << std::endl;

    return 0;
}
